#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>


#include <hip/device_functions.h>

#define MAIN "_MAIN_:"
#define F1LO "_ODD_:"
#define CTRL "_CRTL_:"

#define ARRAY_SIZE 100000
#define TILE_WIDTH 256

__device__
inline void SWAP(int32_t *_a,int32_t *_b){int32_t __aux; __aux = *_a; *_a = *_b; *_b = __aux;}



/*###################################
###########    KERNELS    ###########
###################################*/

__global__
void odd_even_sort_kernel(int32_t * arr_d, int32_t n){
    int32_t t_position = (blockDim.x * blockIdx.x + threadIdx.x)*2 + 1;// +1 corresponde para evitar el overflow en el 0
    int32_t tid = threadIdx.x*2+1;

        for(int32_t i=0; i<blockDim.x;i++){

        	if ((i&1) && t_position< n-1 && tid < blockDim.x*2-1 ) { // impar
                    if (arr_d[t_position + 1] < arr_d[t_position]) {
                        SWAP(arr_d + t_position, arr_d + t_position + 1);
                    }
                }
                if(!(i&1) && t_position < n && tid < blockDim.x*2){ //par
                    if (arr_d[t_position] < arr_d[t_position-1]) {
                        SWAP(arr_d + t_position, arr_d + t_position - 1);
                    }
                }
                __syncthreads();
        }
}

__global__
void fast_odd_even_sort_kernel(int32_t * arr_d, int32_t n){
    int32_t position = (blockDim.x * blockIdx.x + threadIdx.x)*2 + 1;// +1 corresponde para evitar el overflow en el 0
    int32_t tid = threadIdx.x*2+1;
    __shared__ int32_t sh_arr[2*TILE_WIDTH];
    int32_t bound = blockDim.x*2;
    
    if(position < n){
    	*(sh_arr+tid)=*(arr_d+position);
    	*(sh_arr+tid-1)=*(arr_d+position-1);
    	__syncthreads();
    
    	for(int32_t i=0; i<blockDim.x;i++){

    		if ((i&1) && position< n-1 && tid < bound-1 ) { // impar
    			if (*(sh_arr+tid + 1) < *(sh_arr+tid)) {
    				SWAP(sh_arr + tid, sh_arr + tid + 1);
                }
            }

    		if(!(i&1) && position < n && tid < bound){ //par
                if (*(sh_arr+tid) < *(sh_arr+tid-1)) {
                    SWAP(sh_arr + tid, sh_arr + tid - 1);
                }
            }
            __syncthreads();
    	}

    	*(arr_d+position) = *(sh_arr+tid);
    	*(arr_d+position-1) = *(sh_arr+tid-1);
    }
}


/*##########################################
###########    HOST FUNCTIONS    ###########
##########################################*/


__host__
void odd_even_sort(int32_t * arr, int32_t n){
	int32_t *cuda_d;
	dim3 dimGrid ((uint)((ARRAY_SIZE / TILE_WIDTH)+1), 1, 1);
	dim3 dimBlock (TILE_WIDTH, 1, 1);
	hipError_t err;
	hipEvent_t start, stop;
	float mili;

	err = hipMalloc((void**)&cuda_d, sizeof(int32_t)*ARRAY_SIZE);
	if( err != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); // best definition
		exit(EXIT_FAILURE);
	}
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(cuda_d, arr, sizeof(int32_t)*ARRAY_SIZE, hipMemcpyHostToDevice);

	int32_t j_limit = n*2/TILE_WIDTH;
	int32_t *p_cuda;
	int32_t size;

	printf("%s ordenando..\n",F1LO);
    hipEventRecord(start);
	for(int32_t j=0;j<j_limit;j++){
		p_cuda = cuda_d + (j&1) * TILE_WIDTH;
		size = n - (j&1) * TILE_WIDTH;
		odd_even_sort_kernel<<<dimGrid, dimBlock>>>(p_cuda, size);
	}
	hipEventRecord(stop);
	//cudaDeviceSynchronize();
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mili, start, stop);
	hipMemcpy(arr, cuda_d, sizeof(int32_t)*ARRAY_SIZE, hipMemcpyDeviceToHost);

	printf("%s terminanding.. time: %f s\n", F1LO, mili/1000);
	hipFree(cuda_d);
}


__host__
void fast_odd_even_sort(int32_t * arr, int32_t n){
    int32_t *cuda_d;
    //float tile = TILE_WIDTH , size_t = ARRAY_SIZE;
    //dim3 dimGrid ((uint)ceil(size_t/tile), 1, 1);
    dim3 dimGrid ((uint)((ARRAY_SIZE / TILE_WIDTH)+1), 1, 1);
    dim3 dimBlock (TILE_WIDTH, 1, 1);
    hipError_t err;
    hipEvent_t start, stop;
    float mili;

    err = hipMalloc(&cuda_d, sizeof(int32_t)*ARRAY_SIZE);
    if( err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); // best definition
        exit(EXIT_FAILURE);
    }
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(cuda_d, arr, sizeof(int32_t)*ARRAY_SIZE, hipMemcpyHostToDevice);

    int32_t j_limit = n*2/TILE_WIDTH;
    int32_t *p_cuda;
    int32_t size;

    printf("%s ordenando..\n",F1LO);
    hipEventRecord(start);
    for(int32_t j=0;j<j_limit;j++){
        p_cuda = cuda_d + (j&1) * TILE_WIDTH;
        size = n - (j&1) * TILE_WIDTH;
        fast_odd_even_sort_kernel<<<dimGrid, dimBlock>>>(p_cuda, size);
    }
    hipEventRecord(stop);
    //cudaDeviceSynchronize();
    hipEventSynchronize(stop);
    hipEventElapsedTime(&mili, start, stop);
    hipMemcpy(arr, cuda_d, sizeof(int32_t)*ARRAY_SIZE, hipMemcpyDeviceToHost);

    printf("%s terminanding.. time: %f s\n", F1LO, mili/1000);
    hipFree(cuda_d);
}


__host__
int control(int32_t *arr, int32_t n){
  for(int32_t i=1; i<n; i++){
    if(arr[i-1] > arr[i]){
        printf("%s I = %d\n", CTRL, i);
        return 1;
    }
  }
  return 0;
}


/*###################################
###########      MAIN     ###########
###################################*/


int main( int argc, char *argv[] ){
    int32_t *arr;
    
    arr = (int32_t*)malloc(sizeof(int32_t)*ARRAY_SIZE);
    
    printf("array size: %d tile: %d\n",ARRAY_SIZE, TILE_WIDTH);

    printf("####   SORT WHIT GLOBAL MEMORY   ####\n" );
    for (int i = 0; i < ARRAY_SIZE; i++) {
        arr[i] = rand()%1000;
      //  printf("%d ", arr[i]);
    }
    printf("\n");

    if(control(arr, ARRAY_SIZE)) printf("%s desordenado!! \n",MAIN);
    else printf("%s ok!! \n",MAIN);  
        
    odd_even_sort(arr,ARRAY_SIZE);

    if(control(arr, ARRAY_SIZE)) printf("%s desordenado!! \n",MAIN);
    else printf("%s ok!! \n" ,MAIN);


    printf("####   SORT WHIT SHARED MEMORY   ####\n" );
    for (int i = 0; i < ARRAY_SIZE; i++) {
        arr[i] = rand()%1000;
      //  printf("%d ", arr[i]);
    }
    printf("\n");

    if(control(arr, ARRAY_SIZE)) printf("%s desordenado!! \n",MAIN);
    else printf("%s ok!! \n",MAIN);

    fast_odd_even_sort(arr,ARRAY_SIZE);

    if(control(arr, ARRAY_SIZE)) printf("%s desordenado!! \n",MAIN);
    else printf("%s ok!! \n" ,MAIN);

    free(arr);
    printf("\n");

    return 0;
}
