#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>


#include <hip/device_functions.h>

#define MAIN "_MAIN_:"
#define F1LO "_ODD_:"
#define CTRL "_CRTL_:"

#define ARRAY_SIZE 53457
#define TILE_WIDTH 1024

__device__
inline void SWAP(int32_t *_a,int32_t *_b){int32_t __aux; __aux = *_a; *_a = *_b; *_b = __aux;}


__global__
void odd_even_sort_kernel(int32_t * arr_d, int32_t n){
    int32_t position = (blockDim.x * blockIdx.x + threadIdx.x)*2 + 1;// +1 corresponde para evitar el overflow en el 0
    int32_t tid = threadIdx.x*2+1;
    int32_t t_position;
    t_position = position;


        for(int32_t i=0; i<blockDim.x;i++){

        	if ((i&1) && t_position< n-1 && tid < blockDim.x*2-1 ) { // impar
                    if (arr_d[t_position + 1] < arr_d[t_position]) {
                        SWAP(arr_d + t_position, arr_d + t_position + 1);
                    }
                }
                if(!(i&1) && t_position < n && tid < blockDim.x*2){ //par
                    if (arr_d[t_position] < arr_d[t_position-1]) {
                        SWAP(arr_d + t_position, arr_d + t_position - 1);
                    }
                }
                __syncthreads();
        }
}

__host__
void odd_even_sort(int32_t * arr, int32_t n){
	int32_t *cuda_d;
	dim3 dimGrid ((uint)((ARRAY_SIZE / TILE_WIDTH)+1), 1, 1);
	dim3 dimBlock (TILE_WIDTH-1, 1, 1);
	hipError_t err;
	hipEvent_t start, stop;
	float mili;

	err = hipMalloc(&cuda_d, sizeof(int32_t)*ARRAY_SIZE);
	if( err != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); // best definition
		exit(EXIT_FAILURE);
	}
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(cuda_d, arr, sizeof(int32_t)*ARRAY_SIZE, hipMemcpyHostToDevice);

	int32_t j_limit = n*2/TILE_WIDTH;
	int32_t *p_cuda;
	int32_t size;

	printf("%s ordenando..\n",F1LO);
	for(int32_t j=0;j<j_limit;j++){
		p_cuda = cuda_d + (j&1) * TILE_WIDTH;
		size = n - (j&1) * TILE_WIDTH;
		if(j==0)
			hipEventRecord(start);
		odd_even_sort_kernel<<<dimGrid, dimBlock>>>(p_cuda, size);
	}
	hipEventRecord(stop);
	//cudaDeviceSynchronize();
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mili, start, stop);
	hipMemcpy(arr, cuda_d, sizeof(int32_t)*ARRAY_SIZE, hipMemcpyDeviceToHost);

	printf("%s terminanding.. time: %f s\n", F1LO, mili/1000);
	hipFree(cuda_d);
}

__host__
int control(int32_t *arr, int32_t n){
  for(int32_t i=1; i<n; i++){
    if(arr[i-1] > arr[i]){
        printf("%s I = %d\n", CTRL, i);
        return 1;
    }
  }
  return 0;
}

int main( int argc, char *argv[] ){
    int32_t arr[ARRAY_SIZE];



    for (int i = 0; i < ARRAY_SIZE; i++) {
        arr[i] = rand()%1000;
      //  printf("%d ", arr[i]);
    }
    printf("\n");

    if(control(arr, ARRAY_SIZE)) printf("%s desordenado!! \n",MAIN);
    else printf("%s ok!! \n",MAIN);

    odd_even_sort(arr,ARRAY_SIZE);

    if(control(arr, ARRAY_SIZE)) printf("%s desordenado!! \n",MAIN);
    else printf("%s ok!! \n" ,MAIN);
    /*for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%d ", arr[i]);
    }
    */
    printf("\n");

    return 0;
}
